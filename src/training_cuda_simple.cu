#include <rl_tools/operations/hip/hip_runtime.h>
#include <rl_tools/nn/operations_cuda.h>

namespace rlt = RL_TOOLS_NAMESPACE_WRAPPER ::rl_tools;

#include <learning_to_fly/simulator/operations_cpu.h>

#include <iostream>
#include <chrono>

using T = float;
using DEVICE = rlt::devices::DefaultCUDA;
using TI = typename DEVICE::index_t;

int main(void){
    std::cout << "Starting CUDA Training..." << std::endl;
    
    DEVICE device;
    
    // Simple CUDA training placeholder
    std::cout << "CUDA device initialized successfully!" << std::endl;
    
    auto start = std::chrono::high_resolution_clock::now();
    
    // Placeholder for training logic
    for(int i = 0; i < 1000; i++) {
        if(i % 100 == 0) {
            std::cout << "CUDA Training step: " << i << std::endl;
        }
        // TODO: Add actual training logic here
    }
    
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::duration<double>>(end - start).count();
    
    std::cout << "CUDA Training completed in: " << duration << "s" << std::endl;
    
    return 0;
}
